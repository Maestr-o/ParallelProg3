﻿#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"


#define CUDA_CHECK_MALLOC if (cudaStatus != hipSuccess) {\
  fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));\
  return;\
}

#define CUDA_CHECK_MEMCPY if (cudaStatus != hipSuccess) {\
  fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));\
  return;\
}

#define CUDA_CHECK_KERNEL if (cudaStatus != hipSuccess) {\
  fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));\
  return;\
}

void calc(int n);
int* load_primes_from_file(const char* filename, int* size);
__global__ void kernel(int* primes, int* size, int* res, int* n);

int main() {
  int n;
  printf("Enter N: ");
  if (scanf("%d", &n) != 1) {
    printf("Error\n");
    return 0;
  }

  hipEvent_t start, stop;
  float elapsedTime;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  
  calc(n);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("Elapsed time: %.3f\n", elapsedTime / 1000.0);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  return 0;
}

void calc(int n) {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  hipError_t cudaStatus;
  int size;
  int* primes = load_primes_from_file("primes.txt", &size);
  int num_threads = prop.maxThreadsPerBlock;
  int num_blocks = (size + num_threads - 1) / num_threads;

  int* dev_primes;
  int* dev_size;
  int* dev_res;
  int* dev_n;
  int* res = (int*)calloc(size, sizeof(int));
  if (res == NULL) {
    printf("Error allocate memory\n");
    return;
  }

  cudaStatus = hipMalloc((void**)&dev_n, sizeof(int));
  CUDA_CHECK_MALLOC
  cudaStatus = hipMemcpy(dev_n, &n, sizeof(int), hipMemcpyHostToDevice);
  CUDA_CHECK_MEMCPY
  
  cudaStatus = hipMalloc((void**)&dev_size, sizeof(int));
  CUDA_CHECK_MALLOC
  cudaStatus = hipMemcpy(dev_size, &size, sizeof(int), hipMemcpyHostToDevice);
  CUDA_CHECK_MEMCPY

  cudaStatus = hipMalloc((void**)&dev_primes, sizeof(int) * size);
  CUDA_CHECK_MALLOC
  cudaStatus = hipMemcpy(dev_primes, primes, sizeof(int) * size,
                          hipMemcpyHostToDevice);
  CUDA_CHECK_MEMCPY

  cudaStatus = hipMalloc((void**)&dev_res, sizeof(int) * size);
  CUDA_CHECK_MALLOC

  kernel<<<num_blocks, num_threads>>>(dev_primes, dev_size, dev_res, dev_n);
  CUDA_CHECK_KERNEL

  cudaStatus = hipMemcpy(res, dev_res, sizeof(int) * size,
                          hipMemcpyDeviceToHost);
  CUDA_CHECK_MEMCPY

  for (int i = 0; i < size; i++) {
    if (res[i] > n) {
      printf(
          "Pair 1: %d, %d (mid: %d)\nPair 2: %d, %d (mid: %d)\nDiff: %d\n", primes[i],
             primes[i + 1], (primes[i + 1] + primes[i]) / 2, primes[i + 2],
             primes[i + 3], (primes[i + 3] + primes[i + 2]) / 2, res[i]);
      break;
    }
  }

  free(primes);
  hipFree(dev_res);
  hipFree(dev_primes);
  hipFree(dev_size);
  hipFree(dev_n);
}

int* load_primes_from_file(const char* filename, int* size) {
  FILE* file = fopen(filename, "r");
  if (file == NULL) {
    perror("Error opening file");
    return NULL;
  }

  int capacity = 10;
  int* primes = (int*)malloc(sizeof(int) * capacity);
  if (primes == NULL) {
    perror("Memory allocation error");
    fclose(file);
    return NULL;
  }

  int count = 0;
  int num;

  while (fscanf(file, "%d", &num) == 1) {
    if (count == capacity) {
      capacity *= 2;
      int* new_primes = (int*)realloc(primes, sizeof(int) * capacity);
      if (new_primes == NULL) {
        perror("Memory reallocation error");
        free(primes);
        fclose(file);
        return NULL;
      }
      primes = new_primes;
    }
    primes[count++] = num;
  }

  fclose(file);

  int* resized_primes = (int*)realloc(primes, sizeof(int) * count);
  if (resized_primes == NULL) {
    perror("Memory reallocation error");
    free(primes);
    return NULL;
  }

  *size = count;
  return resized_primes;
}

__global__ void kernel(int* primes, int* size, int* res, int* n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid > *size - 3) return;
  int diff = (primes[tid + 3] + primes[tid + 2]) / 2 -
      (primes[tid + 1] + primes[tid]) / 2;
  if (primes[tid + 3] - primes[tid + 2] == 2 &&
      primes[tid + 1] - primes[tid] == 2 &&
      (diff > *n)) {
    res[tid] = diff;
  }
}
