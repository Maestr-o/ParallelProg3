﻿#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include "hip/hip_runtime.h"


#define CUDA_CHECK_MALLOC if (cudaStatus != hipSuccess) {\
  fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));\
  return;\
}

#define CUDA_CHECK_MEMCPY if (cudaStatus != hipSuccess) {\
  fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));\
  return;\
}

#define CUDA_CHECK_KERNEL if (cudaStatus != hipSuccess) {\
  fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));\
  return;\
}

void calc(int n);
bool is_prime(int num);
int* generate_primes(int N, int* length);
__global__ void kernel(int* primes, int* size, int* res, int* n);

int main() {
  int n;
  printf("Enter N: ");
  if (scanf("%d", &n) != 1) {
    printf("Error\n");
    return 0;
  }
  
  clock_t start, end;
  start = clock();
  calc(n);
  end = clock();
  printf("Parallel time: %.3f s", ((double)(end - start)) / CLOCKS_PER_SEC);
  return 0;
}

void calc(int n) {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  hipError_t cudaStatus;

  int size;
  int* primes = generate_primes(pow(10, 7), &size);
  int num_threads = prop.maxThreadsPerBlock;
  int num_blocks = (size + num_threads - 1) / num_threads;

  int* dev_primes;
  int* dev_size;
  int* dev_res;
  int* dev_n;
  int* res = (int*)calloc(size, sizeof(int));
  if (res == NULL) {
    printf("Error allocate memory\n");
    return;
  }

  cudaStatus = hipMalloc((void**)&dev_n, sizeof(int));
  CUDA_CHECK_MALLOC
  cudaStatus = hipMemcpy(dev_n, &n, sizeof(int), hipMemcpyHostToDevice);
  CUDA_CHECK_MEMCPY
  
  cudaStatus = hipMalloc((void**)&dev_size, sizeof(int));
  CUDA_CHECK_MALLOC
  cudaStatus = hipMemcpy(dev_size, &size, sizeof(int), hipMemcpyHostToDevice);
  CUDA_CHECK_MEMCPY

  cudaStatus = hipMalloc((void**)&dev_primes, sizeof(int) * size);
  CUDA_CHECK_MALLOC
  cudaStatus = hipMemcpy(dev_primes, primes, sizeof(int) * size,
                          hipMemcpyHostToDevice);
  CUDA_CHECK_MEMCPY

  cudaStatus = hipMalloc((void**)&dev_res, sizeof(int) * size);
  CUDA_CHECK_MALLOC

  kernel<<<num_blocks, num_threads>>>(dev_primes, dev_size, dev_res, dev_n);
  CUDA_CHECK_KERNEL

  cudaStatus = hipMemcpy(res, dev_res, sizeof(int) * size,
                          hipMemcpyDeviceToHost);
  CUDA_CHECK_MEMCPY

  for (int i = 0; i < size; i++) {
    if (res[i] > n) {
      printf(
          "Pair 1: %d, %d (mid: %d)\nPair 2: %d, %d (mid: %d)\nDiff: %d\n", primes[i],
             primes[i + 1], (primes[i + 1] + primes[i]) / 2, primes[i + 2],
             primes[i + 3], (primes[i + 3] + primes[i + 2]) / 2, res[i]);
      break;
    }
  }

  free(primes);
  hipFree(dev_res);
  hipFree(dev_primes);
  hipFree(dev_size);
  hipFree(dev_n);
}

__global__ void kernel(int* primes, int* size, int* res, int* n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid > *size - 3) return;
  int diff = (primes[tid + 3] + primes[tid + 2]) / 2 -
      (primes[tid + 1] + primes[tid]) / 2;
  if (primes[tid + 3] - primes[tid + 2] == 2 &&
      primes[tid + 1] - primes[tid] == 2 &&
      (diff > *n)) {
    res[tid] = diff;
  }
}

bool is_prime(int num) {
  if (num < 2) return false;
  if (num == 2) return true;
  if (num % 2 == 0) return false;
  for (int i = 3; i * i <= num; i += 2) {
    if (num % i == 0) return false;
  }
  return true;
}

int* generate_primes(int N, int* length) {
  int* primes = (int*)malloc(sizeof(int));
  *length = 0;

  for (int i = 2; i <= N; i++) {
    if (is_prime(i)) {
      (*length)++;
      primes = (int*)realloc(primes, sizeof(int) * (*length));
      primes[(*length) - 1] = i;
    }
  }

  return primes;
}
