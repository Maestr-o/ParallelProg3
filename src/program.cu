﻿#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"


#define CUDA_CHECK_MALLOC if (cudaStatus != hipSuccess) {\
  fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));\
  return;\
}

#define CUDA_CHECK_MEMCPY if (cudaStatus != hipSuccess) {\
  fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));\
  return;\
}

#define CUDA_CHECK_KERNEL if (cudaStatus != hipSuccess) {\
  fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));\
  return;\
}

void calc(int n, int num_blocks, int num_threads);
int* load_primes_from_file(const char* filename, int* size);
__global__ void kernel(int* primes, int* size, int* res, int* n);

int main() {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);

  int n;
  printf("Enter N: ");
  if (scanf("%d", &n) != 1) {
    printf("Error\n");
    return 0;
  }

  int num_threads = prop.maxThreadsPerBlock;
  int num_blocks = (n + num_threads - 1) / num_threads;

  calc(n, num_blocks, num_threads);
  return 0;
}

void calc(int n, int num_blocks, int num_threads) {
  hipError_t cudaStatus;
  int size;
  int* primes = load_primes_from_file("primes.txt", &size);
  int* dev_primes;
  int* dev_size;
  int* dev_res;
  int* dev_n;
  int* res = (int*)calloc(size, sizeof(int));
  if (res == NULL) {
    printf("Error allocate memory\n");
    return;
  }

  cudaStatus = hipMalloc((void**)&dev_n, sizeof(int));
  CUDA_CHECK_MALLOC
  cudaStatus = hipMemcpy(dev_n, &n, sizeof(int), hipMemcpyHostToDevice);
  CUDA_CHECK_MEMCPY
  
  cudaStatus = hipMalloc((void**)&dev_size, sizeof(int));
  CUDA_CHECK_MALLOC
  cudaStatus = hipMemcpy(dev_size, &size, sizeof(int), hipMemcpyHostToDevice);
  CUDA_CHECK_MEMCPY

  cudaStatus = hipMalloc((void**)&dev_primes, sizeof(int) * size);
  CUDA_CHECK_MALLOC
  cudaStatus = hipMemcpy(dev_primes, primes, sizeof(int) * size,
                          hipMemcpyHostToDevice);
  CUDA_CHECK_MEMCPY

  cudaStatus = hipMalloc((void**)&dev_res, sizeof(int) * size);
  CUDA_CHECK_MALLOC

  kernel<<<num_blocks, num_threads>>>(dev_primes, dev_size, dev_res, dev_n);
  CUDA_CHECK_KERNEL

  cudaStatus = hipMemcpy(res, dev_res, sizeof(int) * size,
                          hipMemcpyDeviceToHost);
  CUDA_CHECK_MEMCPY

  for (int i = 0; i < size; i++) {
    if (res[i] > n) {
      printf(
          "Pair 1: %d, %d (mid: %d)\nPair 2: %d, %d (mid: %d)\nDiff: %d\n", primes[i],
             primes[i + 1], (primes[i + 1] + primes[i]) / 2, primes[i + 2],
             primes[i + 3], (primes[i + 3] + primes[i + 2]) / 2, res[i]);
      break;
    }
  }

  hipFree(dev_res);
  hipFree(dev_primes);
  hipFree(dev_size);
  hipFree(dev_n);
}

int* load_primes_from_file(const char* filename, int* size) {
  FILE* file = fopen(filename, "r");
  if (file == NULL) {
    perror("Error opening file");
    return NULL;
  }

  int capacity = 10;
  int* primes = (int*)malloc(sizeof(int) * capacity);
  if (primes == NULL) {
    perror("Memory allocation error");
    fclose(file);
    return NULL;
  }

  int count = 0;
  int num;

  while (fscanf(file, "%d", &num) == 1) {
    if (count == capacity) {
      capacity *= 2;
      int* new_primes = (int*)realloc(primes, sizeof(int) * capacity);
      if (new_primes == NULL) {
        perror("Memory reallocation error");
        free(primes);
        fclose(file);
        return NULL;
      }
      primes = new_primes;
    }
    primes[count++] = num;
  }

  fclose(file);

  int* resized_primes = (int*)realloc(primes, sizeof(int) * count);
  if (resized_primes == NULL) {
    perror("Memory reallocation error");
    free(primes);
    return NULL;
  }

  *size = count;
  return resized_primes;
}

__global__ void kernel(int* primes, int* size, int* res, int* n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid > *size - 3) return;
  int diff = (primes[tid + 3] + primes[tid + 2]) / 2 -
      (primes[tid + 1] + primes[tid]) / 2;
  if (primes[tid + 3] - primes[tid + 2] == 2 &&
      primes[tid + 1] - primes[tid] == 2 &&
      (diff > *n)) {
    res[tid] = diff;
  }
}
