﻿#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <omp.h>

#include "hip/hip_runtime.h"


#define CUDA_CHECK_MALLOC                      \
  if (cudaStatus != hipSuccess) {             \
    fprintf(stderr, "hipMalloc failed: %s\n", \
            hipGetErrorString(cudaStatus));   \
    return;                                    \
  }

#define CUDA_CHECK_MEMCPY                      \
  if (cudaStatus != hipSuccess) {             \
    fprintf(stderr, "hipMemcpy failed: %s\n", \
            hipGetErrorString(cudaStatus));   \
    return;                                    \
  }

#define CUDA_CHECK_KERNEL                         \
  if (cudaStatus != hipSuccess) {                \
    fprintf(stderr, "Kernel launch failed: %s\n", \
            hipGetErrorString(cudaStatus));      \
    return;                                       \
  }

void calc(int n_primes, int n);
bool is_prime(int num);
int* generate_primes(int N, int* length);
__global__ void kernel(int* primes, int* size, int* res, int* n);

int main() {
  int n, n_primes;
  printf("Enter N: ");
  if (scanf("%d", &n) != 1) {
    printf("Error\n");
    return 0;
  }
  printf("Enter the maximum number of primes in array: ");
  if (scanf("%d", &n_primes) != 1) {
    printf("Error\n");
    return 0;
  }

  double start, end;
  start = omp_get_wtime();
  calc(n_primes, n);
  end = omp_get_wtime();
  printf("Time: %.3f sec\n", (end - start));
  return 0;
}

void calc(int n_primes, int n) {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  hipError_t cudaStatus;

  int size;
  int* primes = generate_primes(n_primes, &size);
  int num_threads = prop.maxThreadsPerBlock;
  int num_blocks = (size + num_threads - 1) / num_threads;

  int* dev_primes;
  int* dev_size;
  int* dev_res;
  int* dev_n;
  int* res = (int*)calloc(size, sizeof(int));
  if (res == NULL) {
    printf("Error allocate memory\n");
    return;
  }

  cudaStatus = hipMalloc((void**)&dev_n, sizeof(int));
  CUDA_CHECK_MALLOC
  cudaStatus = hipMemcpy(dev_n, &n, sizeof(int), hipMemcpyHostToDevice);
  CUDA_CHECK_MEMCPY

  cudaStatus = hipMalloc((void**)&dev_size, sizeof(int));
  CUDA_CHECK_MALLOC
  cudaStatus =
      hipMemcpy(dev_size, &size, sizeof(int), hipMemcpyHostToDevice);
  CUDA_CHECK_MEMCPY

  cudaStatus = hipMalloc((void**)&dev_primes, sizeof(int) * size);
  CUDA_CHECK_MALLOC
  cudaStatus = hipMemcpy(dev_primes, primes, sizeof(int) * size,
                          hipMemcpyHostToDevice);
  CUDA_CHECK_MEMCPY

  cudaStatus = hipMalloc((void**)&dev_res, sizeof(int) * size);
  CUDA_CHECK_MALLOC

  kernel<<<num_blocks, num_threads>>>(dev_primes, dev_size, dev_res, dev_n);
  CUDA_CHECK_KERNEL

  cudaStatus =
      hipMemcpy(res, dev_res, sizeof(int) * size, hipMemcpyDeviceToHost);
  CUDA_CHECK_MEMCPY

  bool is_answer_found = false;
  for (int i = 0; i < size; i++) {
    if (res[i] > n) {
      if (primes[i] > 0 && primes[i + 1] > 0 && primes[i + 2] > 0 && primes[i + 3] > 0) {
        printf("Pair 1: %d, %d (mid: %d)\nPair 2: %d, %d (mid: %d)\nDiff: %d\n",
             primes[i], primes[i + 1], (primes[i + 1] + primes[i]) / 2,
             primes[i + 2], primes[i + 3], (primes[i + 3] + primes[i + 2]) / 2,
             res[i]);
        is_answer_found = true;
        break;
      } else
        break;
    }
  }
  if (!is_answer_found)
    printf(
        "Answer wasn't found. Please increase the maximum number of primes\n");

  free(primes);
  hipFree(dev_res);
  hipFree(dev_primes);
  hipFree(dev_size);
  hipFree(dev_n);
}

__global__ void kernel(int* primes, int* size, int* res, int* n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid > *size - 3) return;
  int diff = (primes[tid + 3] + primes[tid + 2]) / 2 -
              (primes[tid + 1] + primes[tid]) / 2;
  if (primes[tid + 3] - primes[tid + 2] == 2 &&
      primes[tid + 1] - primes[tid] == 2 && (diff > *n)) {
    res[tid] = diff;
  } else
    res[tid] = 0;
}

bool is_prime(int num) {
  if (num < 2) return false;
  for (int i = 2; i <= sqrt(num); i++) {
    if (num % i == 0) return false;
  }
  return true;
}

int* generate_primes(int N, int* length) {
  int* primes = (int*)malloc(sizeof(int));
  *length = 0;

  for (int i = 2; i <= N; i++) {
    if (is_prime(i)) {
      (*length)++;
      primes = (int*)realloc(primes, sizeof(int) * (*length));
      primes[(*length) - 1] = i;
    }
  }

  return primes;
}
